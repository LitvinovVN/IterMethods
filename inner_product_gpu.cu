// nvcc inner_product_gpu.cu -o inner_product_gpu -O3 -dopt=on
// ./inner_product_gpu

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

const int N = 2048*10;

const int nBlockNum = 16;//Open 32 blocks
const int threadnum = 32;//Open 32 threads


/* Three, single-block low-thread reduction vector inner product*/

/* Four, multi-block multi-thread vector inner product */
template <typename T>
__global__ void dot_gpu_3(T *a, T *b, T *c, int n)
{
    __shared__ T aTmp[threadnum];
    const int nThreadIdX = threadIdx.x; //Thread ID index number
    const int nStep = gridDim.x * blockDim.x; // The step length of the step, that is, the number of all threads
    int nTidIdx = blockIdx.x * blockDim.x + threadIdx.x; // The index of the current thread in the global thread

    double dTemp = 0.0;
    while (nTidIdx < n)
    {
        dTemp += a[nTidIdx] * b[nTidIdx];
        nTidIdx += nStep;
    }
    aTmp[nThreadIdX] = dTemp; // Put the inner product in each thread into the shared memory of the corresponding block
    __syncthreads(); // Synchronous operation, that is, wait for the above operations in all threads to be executed

    int i = threadnum / 2;
    while (i != 0)
    {
        if(nThreadIdX < i)
        {
            aTmp[nThreadIdX] += aTmp[nThreadIdX + i];
        }
        __syncthreads(); // Synchronous operation, that is, wait for the above operations in all threads to be executed
        i /= 2;
    }

    if (0 == nThreadIdX)
    {
        c[blockIdx.x] = aTmp[0];
    }

}

int main()
{
    float a[N], b[N];    
    for(int i=0; i<N; ++i) // Assign values ​​to arrays a and b
    {
        a[i] = i * 0.1;
        if(fabs(a[i] < 0.000001))
            b[i] = 0.0;
        else
            b[i] = 1/a[i];
    }

    float *d_a = NULL, *d_b = NULL, *d_c = NULL;
    hipMalloc(&d_a, N *sizeof(float));
    hipMemcpy(d_a, a, N *sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_b, N *sizeof(float));
    hipMemcpy(d_b, b, N *sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_c, sizeof(float) * nBlockNum);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    dot_gpu_3<<< nBlockNum, threadnum >>>(d_a, d_b, d_c, N);
    
    float c_tmp[nBlockNum];
    hipMemcpy(&c_tmp, d_c, nBlockNum * sizeof(float), hipMemcpyDeviceToHost);
    
    float c = 0;
    for (int i=0; i < nBlockNum; ++i)
    {
        c += c_tmp[i];
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Elapsed " << milliseconds << " ms." << std::endl;

    std::cout << c << std::endl;
    printf("--- End of CUDA Inner Productst---\n");
    return 0;
}

