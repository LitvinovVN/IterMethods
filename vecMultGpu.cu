#include "hip/hip_runtime.h"
// Вычисление скалярного произведения на GPU
// nvcc vecMultGpu.cu -o app
// ./app

#include "vector1d.c"


#include <stdio.h>

hipError_t vecMultCuda(const vector1d *v1, const vector1d *v2, double* result);

__global__ void vecMultKernel(vector1d *v1, vector1d *v2, double* result)
{
    int i = threadIdx.x;
    //c[i] = a[i] + b[i];
    printf("GPU v1: Length = %d\n", v1->length);
    printf("GPU v1: Data[0] = %lf\n", v1->data[0]);
    printf("GPU v2: Length = %d\n", v2->length);
    printf("GPU v2: Data[0] = %lf\n", v2->data[0]);

    double res = 0;
    for(size_t i = 0; i < v1->length; i++)
        res += v1->data[i] * v2->data[i];

    *result = res;
}

int main()
{
    size_t size = 10;
    
    vector1d *v1 = vector1d_create(size); 
    vector1d *v2 = vector1d_create(size);

    double k = 0.5;
    int min = 10;
    int max = 20;


    clock_t start, end;
    double cpu_time_used;
        
    vector1d_init_scalar(v1, v2, k, min, max);
    vector1d_print(v1, "v1 = ");
    vector1d_print(v2, "v2 = ");
    
    start = clock();

    // Вычисление скалярного произведения
    double scalar_mult_result = 0;
    hipError_t cudaStatus = vecMultCuda(v1, v2, &scalar_mult_result);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vecMultCuda failed!");
        return 1;
    }
    
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("cpu_time_used = %lf sec\n", cpu_time_used);

    printf("scalar_mult_result = %lf", scalar_mult_result);


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to multiply vectors in parallel.
hipError_t vecMultCuda(const vector1d *v1, const vector1d *v2, double* result)
{
    vector1d *dev_v1 = 0;
    vector1d *dev_v2 = 0;
    double *dev_result = 0;
    hipError_t cudaStatus;

    // Выбор GPU для запуска, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    double *dev_v1_data = 0;
    cudaStatus = hipMalloc((void**)&dev_v1_data, v1->length * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }    
    cudaStatus = hipMalloc((void**)&dev_v1, sizeof(vector1d));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    vector1d v1_dto;
    v1_dto.length = v1->length;
    v1_dto.data = dev_v1_data;


    double *dev_v2_data = 0;
    cudaStatus = hipMalloc((void**)&dev_v2_data, v2->length * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_v2, sizeof(vector1d));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    vector1d v2_dto;
    v2_dto.length = v2->length;
    v2_dto.data = dev_v2_data;

    cudaStatus = hipMalloc((void**)&dev_result, sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_v1_data, v1->data, v1->length * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_v1, &v1_dto, sizeof(vector1d), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_v2_data, v2->data, v2->length * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_v2, &v2_dto, sizeof(vector1d), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vecMultKernel<<<1, 1>>>(dev_v1, dev_v2, dev_result);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vecMultKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(result, dev_result, sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    } // */

Error:
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_result);
    
    return cudaStatus;
}
